#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess) {
        printf("Error getting device count: %s\n", hipGetErrorString(error));
        return -1;
    }
    
    printf("Number of CUDA devices: %d\n", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        printf("\nDevice %d properties:\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total global memory: %zu bytes\n", prop.totalGlobalMem);
        printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
    }
    
    return 0;
}
